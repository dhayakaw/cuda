#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

#define CHECK(call)                                                     \
{                                                                       \
  const hipError_t error = call;                                       \
  if(error!=hipSuccess) {                                              \
    printf("Error: %s:%d, ", __FILE__, __LINE__);                       \
    printf("code: %d, reason: %s\n", error, hipGetErrorString(error)); \
  }                                                                     \
}                                                                       \

void InitialInt(int *ip, int size) {
  for(int i=0; i<size; i++) {
    ip[i]=i;
  }
}

void printMatrix(int *C, const int nx, const int ny) {
  int *ic = C;
  cout << "Matrix: " << nx << ", " << ny << endl;
  for (int iy=0; iy<ny; iy++) {
    for (int ix=0; ix<nx; ix++) {
      printf("%3d", ic[ix]);
    }
    ic+=nx;
    cout << endl;
  }
  cout << endl;
}

__global__ void printThreadIndex(int *A, const int nx, const int ny) {
  int ix = threadIdx.x+blockIdx.x*blockDim.x;
  int iy = threadIdx.y+blockIdx.y*blockDim.y;
  unsigned int idx = iy*nx+ix;

  printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}

int main(int argc, char **argv) {
  cout << argv[0] << " Starting..." << endl;

  // get device information
  int dev = 0;
  hipDeviceProp_t deviceProp;
  CHECK(hipGetDeviceProperties(&deviceProp, dev));
  std::cout << "Using Device " << dev << ": " << deviceProp.name << std::endl;
  CHECK(hipSetDevice(dev));

  // set matrix dimension
  int nx=8;
  int ny=6;
  int nxy=nx*ny;
  int nBytes = nxy*sizeof(float);

  // malloc host memory
  int *h_A;
  h_A = (int *)malloc(nBytes);

  // initialize host matrix with integer
  InitialInt(h_A, nxy);
  printMatrix(h_A, nx, ny);

  // malloc device memory
  int *d_MatA;
  hipMalloc((void **) &d_MatA, nBytes);

  // transfer data from host to device
  hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);

  // setup execution configuration
  dim3 block(4,2);
  dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

  // invoke the kernel
  printThreadIndex <<<grid, block>>> (d_MatA, nx, ny);

  // free host and device memory
  hipFree(d_MatA);
  free(h_A);

  // reset device
  hipDeviceReset();

  return 0;
}
