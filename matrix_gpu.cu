#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define MATRIX_SIZE 1024
#define BLOCK_SIZE 16

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC);

int main(int argc, char** argv){
unsigned int matrixSize = sizeof(unsigned int) * MATRIX_SIZE * MATRIX_SIZE;

  int* hMatrixA;
  int* hMatrixB;
  int* hMatrixC;
  hMatrixA = (int*)malloc(matrixSize);
  hMatrixB = (int*)malloc(matrixSize);

  unsigned int col_idx, row_idx;
  for (col_idx = 0; col_idx < MATRIX_SIZE; col_idx++){
      for (row_idx = 0; row_idx < MATRIX_SIZE; row_idx++){
          hMatrixA[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024*1024);
          hMatrixB[col_idx * MATRIX_SIZE + row_idx] = rand() % (1024*1024);
      }
  }

  int* dMatrixA;
  int* dMatrixB;
  int* dMatrixC;

  checkCudaErrors(hipMalloc((void**)&dMatrixA, matrixSize));
  checkCudaErrors(hipMemcpy(dMatrixA, hMatrixA, matrixSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void**)&dMatrixB, matrixSize));
  checkCudaErrors(hipMemcpy(dMatrixB, hMatrixB, matrixSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void**)&dMatrixC, matrixSize));

  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(MATRIX_SIZE/BLOCK_SIZE, MATRIX_SIZE/BLOCK_SIZE);

  hipEvent_t start;
  hipEvent_t stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors(hipEventRecord(start, NULL)); //start

  matrixMul<<<grid, block>>>(dMatrixA, dMatrixB, dMatrixC);
  hipDeviceSynchronize();

  hMatrixC = (int*)malloc(matrixSize);
  checkCudaErrors(hipMemcpy(hMatrixC, dMatrixC, matrixSize, hipMemcpyDeviceToHost));

  checkCudaErrors(hipEventRecord(stop, NULL));
  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  printf("Processing time: %f (msec)\n", msecTotal);

  free(hMatrixA);
  free(hMatrixB);
  free(hMatrixC);
  checkCudaErrors(hipFree(dMatrixA));
  checkCudaErrors(hipFree(dMatrixB));
  checkCudaErrors(hipFree(dMatrixC));

  hipDeviceReset();
  exit(1);
}

__global__ void
matrixMul(int* inMatrixA, int* inMatrixB, int* inMatrixC){
  unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int scan_idx;
  unsigned int target = 0;

 for (scan_idx = 0; scan_idx < MATRIX_SIZE; scan_idx++) {
   target +=inMatrixA[col_idx * MATRIX_SIZE + scan_idx] * inMatrixB[scan_idx * MATRIX_SIZE + row_idx];
   __syncthreads();
 }
 inMatrixC[col_idx * MATRIX_SIZE + row_idx] = target;
}
